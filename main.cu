
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <cstdio>

__global__ void whoami(void) {
    int block_id = 
        blockIdx.x +
        blockIdx.y * gridDim.x +
        blockIdx.x * gridDim.x * gridDim.y;

    int block_offset = 
        block_id * 
        blockDim.x * blockDim.y * blockDim.z;

    int thread_offset =
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;

    printf("%04d\n", id); 
}

int main(int argc, char const *argv[])
{  
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;
    
    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    std::cout << "bl/g: " << blocks_per_grid << std::endl;
    std::cout << "th/g: " << threads_per_block << std::endl;
    std::cout << "total threads: " << blocks_per_grid * threads_per_block << std::endl;


    dim3 cuBPG(b_x, b_y, b_z);
    dim3 cuTPB(t_x, t_y, t_z);


    whoami<<<cuBPG, cuTPB>>>();
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
