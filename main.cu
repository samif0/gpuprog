
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

int main(int argc, char const *argv[])
{  
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;
    
    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    std::cout << "bl/g: " << blocks_per_grid << std::endl;
    std::cout << "th/g: " << threads_per_block << std::endl;
    std::cout << "total threads: " << blocks_per_grid * threads_per_block << std::endl;
    return EXIT_SUCCESS;
}
